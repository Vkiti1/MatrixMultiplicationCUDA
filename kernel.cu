#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <stdio.h>
#include<iostream>
#include<cstdlib>
#include<chrono>
#include<assert.h>

using namespace std;
//izracun 
__global__ void multiply(int* a, int* b, int* c, int n) {
	//izracun retka i stupca za svaki thread
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	//provjera granica
	if ((row < n) && (col < n)) {
		int tmp = 0;
		for (int i = 0; i < n; i++) {
			tmp += a[row * n + i] * b[i * n + col];
		}

		c[row * n + col] = tmp;
	}
}

//izvodenje na procesoru
void runOnHost(int* a, int* b, int* c, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			for (int k = 0; k < n; k++) {
				c[i * n + j] += a[i * n + k] * b[k * n + j];
			}
		}
	}
}

//provjera rezultata
void verify(int* a, int* b, int* c, int n) {

	cout << "Provjera rezultata: \n";

	int tmp;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			tmp = 0;
			for (int k = 0; k < n; k++) {
				tmp += a[i * n + k] * b[k * n + j];
			}
			if (c[i * n + j] != tmp)
				cout << "nije dobro" << endl;
		}
	}
}

void init_matrices(int* m, int n) {
	for (int i = 0; i < n * n; i++) {
		m[i] = rand() % 10;
	}
}

int main() {
	int N = 1 << 10; //dimenzije matrica, 1024x1024, GPU postaje br�i na N = 1 << 6
	size_t bytes = N * N * sizeof(int);

	//host pointeri
	int* h_a, * h_b, * h_c, * h_c_cpu;//rezultantna matrica za izracun na hostu

	//device pointeri
	int* d_a, * d_b, * d_c;

	//alokacija host memorije
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);
	h_c_cpu = (int*)malloc(bytes);

	//alokacija device memorije
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	//inicijalizacija matrica
	init_matrices(h_a, N);
	init_matrices(h_b, N);

	//kopiranje matrica na GPU
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	int BLOCK_SIZE = 16;
	int GRID_SIZE = (int)ceil(N / BLOCK_SIZE);

	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	//pocetak mjerenja za GPU
	chrono::steady_clock::time_point begin = chrono::steady_clock::now();

	//launch kernel
	multiply << <grid, threads >> > (d_a, d_b, d_c, N);
	hipDeviceSynchronize();



	//kopiraj rezultat sa GPU-a na host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	//zavrsetak mjerenja za GPU
	chrono::steady_clock::time_point end = chrono::steady_clock::now();

	cout << "Vrijeme za izracun na GPU = " << chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << " mikrosekundi" << endl;

	//pocetak mjerenja za CPU
	chrono::steady_clock::time_point beginCPU = chrono::steady_clock::now();

	//izracun na CPU
	runOnHost(h_a, h_b, h_c_cpu, N);

	//kraj mjerenja za CPU
	chrono::steady_clock::time_point endCPU = chrono::steady_clock::now();

	cout << "Vrijeme za izracun na CPU = " << chrono::duration_cast<std::chrono::milliseconds>(endCPU - beginCPU).count() << " mikrosekundi" << endl;

	verify(h_a, h_b, h_c, N);

	cout <<"\neverything works" << endl;

	//ispis dobivene matrice
	/*for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << h_c[i * N + j] << " ";
		}
		cout << endl;
	}*/

	/*for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << h_c_cpu[i * N + j] << " ";
		}
		cout << endl;
	}*/

	//oslobadanje memorije na hostu
	free(h_a);
	free(h_b);
	free(h_c);

	//oslobadanje memorije na GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

